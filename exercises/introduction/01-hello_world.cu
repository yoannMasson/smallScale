#include "hip/hip_runtime.h"
// Copyright 2014, Cranfield University
// All rights reserved
// Author: Michał Czapiński (mczapinski@gmail.com)
//
// Demonstrates the most basic CUDA concepts on the example
// of single precision AXPY operation.
// AXPY stands for y = y + alpha * x, where x, and y are vectors.

#include <iostream>

#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers

// With this implementation and 256 threads per block, works only for up to 16M. Why?
const int N = 15 * 1024 * 1024;
const dim3 BLOCK_DIM = 256;

// Simple CPU implementation of a single precision AXPY operation.
void CpuSaxpy(int n, float alpha, const float* x, float* y) {
  for (int i = 0; i < n; ++i) {
    y[i] += alpha * x[i];
  }
}

// GPU implementation of AXPY operation - one CUDA thread per vector element.
__global__ void GpuSaxpy(int n, float alpha, const float* x, float* y) {
  // TODO Calculate the index of the vector element updated by this thread.
  // Assume 1D grid of blocks.
  int idx = 0;

  // TODO Make sure that no threads access memory outside the allocated area.

}

// GPU implementation of AXPY operation - CUDA thread updates multiple vector elements.
__global__ void GpuSaxpyMulti(int n, float alpha, const float* x, float* y) {
  // TODO(later) Implement CUDA kernel where threads update more than one vector element.
  // Assume 1D grid of blocks.

  // TODO(later) Check if it's faster than the original implementation.
}

int main(int argc, char** argv) {

// ----------------------- Host memory initialisation ----------------------- //

  float* h_x = new float[N];
  float* h_y = new float[N];

  // Initialise vectors on the CPU.
  std::fill_n(h_x, N, 1.0f);  // Vector of ones
  for (int i = 0; i < N; ++i) {
    h_y[i] = 0.33f * (i + 1);
  }

// ---------------------- Device memory initialisation ---------------------- //

  // TODO Allocate global memory on the GPU. Each vector should have N float elements.
  float* d_x = 0;
  float* d_y = 0;
  // hipMalloc(...)

  // TODO Copy vectors from the host (CPU) to the device (GPU).
  // hipMemcpy(...)

// --------------------- Calculations for CPU implementation ---------------- //

  // Create the CUDA SDK timer.
  StopWatchInterface* timer = 0;
  sdkCreateTimer(&timer);

  timer->start();
  CpuSaxpy(N, 0.25f, h_x, h_y);  // y = y + 0.25 * x;
  CpuSaxpy(N, -10.5f, h_x, h_y);  // y = y - 10.5 * x;

  timer->stop();
  std::cout << "CPU time: " << timer->getTime() << " ms." << std::endl;

// --------------------- Calculations for GPU implementation ---------------- //

  // TODO Calculate the number of required thread blocks (one thread per vector element).
  const dim3 GRID_DIM;

  timer->reset();
  timer->start();
  // TODO Insert the correct kernel invocation parameters.
//  GpuSaxpy<<<...>>>(N, 0.25f, d_x, d_y);
//  GpuSaxpy<<<...>>>(N, -10.5f, d_x, d_y);

  // This should work as well.
//  GpuSaxpyMulti<<<...>>>(N, 0.25f, d_x, d_y);
//  GpuSaxpyMulti<<<...>>>(N, -10.5f, d_x, d_y);

  // Kernel calls are asynchronous with respect to the host, i.e. control is returned to
  // the CPU immediately. It is possible that the second operation is submitted _before_
  // the first one is completed. However, CUDA driver will ensure that they will be
  // completed in FIFO order, one at a time.

  // CPU has to explicitly wait for the device to complete
  // in order to get meaningful time measurement.
  checkCudaErrors(hipDeviceSynchronize());
  timer->stop();
  std::cout << "GPU time: " << timer->getTime() << " ms." << std::endl;

  // TODO Download the resulting vector d_y from the device and store it in h_x.
  // hipMemcpy(...)

  // hipMemcpy is synchronous, i.e. it will wait for any computation on the GPU to
  // complete before any data is copied (as if hipDeviceSynchronize() was called before).

  // Now let's check if the results are the same.
  float diff = 0.0f;
  for (int i = 0; i < N; ++i) {
    diff = std::max(diff, std::abs(h_x[i] - h_y[i]));
  }
  std::cout << "Max diff = " << diff << std::endl;  // Should be (very close to) zero.

// ------------------------------- Cleaning up ------------------------------ //

  delete timer;
  delete[] h_x;
  delete[] h_y;

  // TODO Don't forget to free host and device memory!
  // hipFree(...)

  return 0;
}
