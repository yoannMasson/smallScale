#include "hip/hip_runtime.h"
// Copyright 2014, Cranfield University
// All rights reserved
// Author: Michał Czapiński (mczapinski@gmail.com)
//
// Adds two matrices on the GPU. Matrices are stored in linear memory in row-major order,
// i.e. A[i, j] is stored in i * COLS + j element of the vector.

#include <iostream>

#include <hip/hip_runtime.h>  // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h>  // For CUDA SDK timers

// Matrix dimensions. Can you make these input arguments?
const int ROWS = 4096;
const int COLS = 4096;

// TODO(later) Play a bit with the block size. Is 16x16 setup the fastest possible?
// Note: For meaningful time measurements you need sufficiently large matrix.
const dim3 BLOCK_DIM(16, 16);

// Simple CPU implementation of matrix addition.
void CpuMatrixAdd(int rows, int cols, const float* A, const float* B, float* C) {
  for (int row = 0; row < rows; ++row) {
    for (int col = 0; col < cols; ++col) {
      int idx = row * cols + col;
      C[idx] = A[idx] + B[idx];
    }
  }
}

// GPU implementation of matrix add using one CUDA thread per vector element.
__global__ void GpuMatrixAdd(int rows, int cols, const float* A, const float* B, float* C) {
  // TODO Calculate indices of matrix elements added by this thread. Assume 2D grid of blocks.
  int col = 0;
  int row = 0;
  // TODO(later) Does it matter if you index rows with x or y dimension of threadIdx and blockIdx?

  // TODO Calculate the element index in the global memory and add the values.
  // TODO Make sure that no threads access memory outside the allocated area.
}

int main(int argc, char** argv) {

// ----------------------- Host memory initialisation ----------------------- //

  float* h_A = new float[ROWS * COLS];
  float* h_B = new float[ROWS * COLS];
  float* h_C = new float[ROWS * COLS];

  srand(time(0));
  for (int row = 0; row < ROWS; ++row) {
    for (int col = 0; col < COLS; ++col) {
      int idx = row * COLS + col;
      h_A[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
      h_B[idx] = 100.0f * static_cast<float>(rand()) / RAND_MAX;
    }
  }

// ---------------------- Device memory initialisation ---------------------- //

  // TODO Allocate global memory on the GPU.
  float *d_A, *d_B, *d_C;

  // TODO Copy matrices from the host (CPU) to the device (GPU).

// ------------------------ Calculations on the CPU ------------------------- //

  // Create the CUDA SDK timer.
  StopWatchInterface* timer = 0;
  sdkCreateTimer(&timer);

  timer->start();
  CpuMatrixAdd(ROWS, COLS, h_A, h_B, h_C);

  timer->stop();
  std::cout << "CPU time: " << timer->getTime() << " ms." << std::endl;

// ------------------------ Calculations on the GPU ------------------------- //

  // TODO Calculate the dimension of the grid of blocks (2D).
  const dim3 GRID_DIM;

  timer->reset();
  timer->start();
  GpuMatrixAdd<<<GRID_DIM, BLOCK_DIM>>>(ROWS, COLS, d_A, d_B, d_C);
  checkCudaErrors(hipDeviceSynchronize());

  timer->stop();
  std::cout << "GPU time: " << timer->getTime() << " ms." << std::endl;

  // TODO Download the resulting matrix d_C from the device and store it in h_A.

  // Now let's check if the results are the same.
  float diff = 0.0f;
  for (int row = 0; row < ROWS; ++row) {
    for (int col = 0; col < COLS; ++col) {
      int idx = row * COLS + col;
      diff = std::max(diff, std::abs(h_A[idx] - h_C[idx]));
    }
  }
  std::cout << "Max diff = " << diff << std::endl;  // Should be (very close to) zero.

// ------------------------------- Cleaning up ------------------------------ //

  delete timer;

  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));

  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
  return 0;
}
